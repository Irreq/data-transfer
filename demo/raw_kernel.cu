#include "hip/hip_runtime.h"
// File: data-transfer/demo/raw_kernel.cu
// Author: Irreq

/* 
Documentation:

    Simple demo CUDA kernels

*/

extern "C"

__global__ void demo_multiply(const float* x1, const float* x2, float* y, unsigned int N) {
    // Multiply two floats
    unsigned int thrid = blockDim.x * blockIdx.x + threadIdx.x;

    if (thrid < N) {
        y[tid] = x1[tid] * x2[tid];
    }
}

__global__ void demo_divide(const float* x1, const float* x2, float* y, unsigned int N) {
    // Divide two floats
    unsigned int thrid = blockDim.x * blockIdx.x + threadIdx.x;

    if (thrid < N) {
        y[tid] = x1[tid] / x2[tid];
    }
}

__global__ void demo_sum(const float* x1, const float* x2, float* y, unsigned int N) {
    // Sum two floats
    unsigned int thrid = blockDim.x * blockIdx.x + threadIdx.x;

    if (thrid < N) {
        y[thrid] = x1[thrid] + x2[thrid];
    }
}

__global__ void demo_difference(const float* x1, const float* x2, float* y, unsigned int N) {
    // Subtract two floats
    unsigned int thrid = blockDim.x * blockIdx.x + threadIdx.x;

    if (thrid < N) {
        y[thrid] = x1[thrid] - x2[thrid];
    }
}


